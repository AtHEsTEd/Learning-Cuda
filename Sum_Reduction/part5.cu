#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include <algorithm>
#include <chrono>

#define SIZE 256

using namespace std::chrono;

// Saves work during last iteration
// Volatile prevents register caching
__device__ void warp_reduce(volatile int* smem_ptr, int t)
{
	smem_ptr[t] += smem_ptr[t + 32];
	smem_ptr[t] += smem_ptr[t + 16];
	smem_ptr[t] += smem_ptr[t + 8];
	smem_ptr[t] += smem_ptr[t + 4];
	smem_ptr[t] += smem_ptr[t + 2];
	smem_ptr[t] += smem_ptr[t + 1];
}


__global__ void sum_reduction(int* vector, int* result_vector)
{
	__shared__ int partial_sum[SIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// Load elements and do first add of reduction
	// Vector will be 2x no of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	partial_sum[threadIdx.x] = vector[i] + vector[i + blockDim.x];
	__syncthreads();

	// Start at 1/2 block stride and halve each time
	for (int s = blockDim.x / 2; s > 32; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x < 32)
	{
		warp_reduce(partial_sum, threadIdx.x);
	}

	if (threadIdx.x == 0)
	{
		result_vector[blockIdx.x] = partial_sum[0];
	}
}


int main()
{
	int n = 1 << 18;
	size_t bytes = n * sizeof(int);

	int* host_v, * host_v_r;
	int* device_v, * device_v_r;

	host_v = (int*)malloc(bytes);
	host_v_r = (int*)malloc(bytes);
	hipMalloc(&device_v, bytes);
	hipMalloc(&device_v_r, bytes);

	std::fill_n(host_v, n, 1);
	hipMemcpy(device_v, host_v, bytes, hipMemcpyHostToDevice);

	const int THREADS = SIZE;
	const int BLOCKS = (int)ceil(n / static_cast<double>(THREADS) / 2);

	auto start = high_resolution_clock::now();
	sum_reduction << < BLOCKS, THREADS >> > (device_v, device_v_r);
	sum_reduction << < 1, THREADS >> > (device_v_r, device_v_r);
	auto end = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(end - start);
	printf("GPU took %d microseconds\n", duration);

	hipMemcpy(host_v_r, device_v_r, bytes, hipMemcpyDeviceToHost);

	printf("%d\n", host_v_r[0]);
	assert(host_v_r[0] == n);

	return 0;
}